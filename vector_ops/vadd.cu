#include "hip/hip_runtime.h"
#include "bits/stdc++.h"
#include "../helper/error.cuh"

using namespace std;

__global__ void vadd(float *a, float *b, float *c, int N) {
  int tid = blockDim.x * blockIdx.x + threadIdx.x;
  if (tid < N) {
    c[tid] = a[tid] + b[tid];
  }
}

int main() {
  int nums_n[] = {128, 512, 1024, 2048, 4096};
  int n_size = sizeof(nums_n) / sizeof(int);

  for (int i = 0; i < n_size; i++) {
    int n = nums_n[i];

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    float *d_a, *d_b, *d_c;
    CHECK(hipMalloc((void **)&d_a, n * sizeof(float)));
    CHECK(hipMalloc((void **)&d_b, n * sizeof(float)));
    CHECK(hipMalloc((void **)&d_c, n * sizeof(float)));

    hipEventRecord(start);
    vadd<<<(n + 256) / 256, 256>>>(d_a, d_b, d_c, n);
    hipEventRecord(stop);

    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("The vector add(size %d) time elapsed: %f ms\n", n, milliseconds);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
  }
}
