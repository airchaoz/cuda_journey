
#include <hip/hip_runtime.h>
#include "bits/stdc++.h"

using namespace std;

__global__ void vadd(float *a, float *b, float *c, int N) {
  int tid = blockDim.x * blockIdx.x + threadIdx.x;
  if (tid < N) {
    c[tid] = a[tid] + b[tid];
  }
}

int main() {
  int nums_n[] = {128, 512, 1024, 2048, 4096};
  int n_size = sizeof(nums_n) / sizeof(int);

  for (int i = 0; i < n_size; i++) {
    int n = nums_n[i];

    float *d_a, *d_b, *d_c;
    hipMalloc((void **)&d_a, n * sizeof(float));
    hipMalloc((void **)&d_a, n * sizeof(float));
    hipMalloc((void **)&d_a, n * sizeof(float));

    vadd<<<(n + 256) / 256, 256>>>(d_a, d_b, d_c, n);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
  }
}
