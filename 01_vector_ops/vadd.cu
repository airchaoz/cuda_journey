#include "hip/hip_runtime.h"
#include "bits/stdc++.h"
#include "../helper/error.cuh"

using namespace std;

#define FLOAT4(x) (reinterpret_cast<float4*>(&(x))[0])

__global__ void vadd(float *a, float *b, float *c, int N) {
  int tid = blockDim.x * blockIdx.x + threadIdx.x;
  if (tid < N) {
    c[tid] = a[tid] + b[tid];
  }
}

__global__ void vadd_f32x4(float *a, float *b, float *c, int N) {
  int tid = 4 * (blockDim.x * blockIdx.x + threadIdx.x);
  if (tid < N) {
    float4 reg_a = FLOAT4(a[tid]);
    float4 reg_b = FLOAT4(a[tid]);
    float4 reg_c = FLOAT4(a[tid]);

    reg_c.x = reg_a.x + reg_b.x;
    reg_c.y = reg_a.y + reg_b.y;
    reg_c.z = reg_a.z + reg_b.z;
    reg_c.w = reg_a.w + reg_b.w;

    FLOAT4(c[tid]) = reg_c;
  }
}

void test_performance(int n) {

  int block_size = 256;
  int grid_size = (n + block_size - 1) / block_size;

  float *h_a = (float *)malloc(n * sizeof(float));
  float *h_b = (float *)malloc(n * sizeof(float));
  float *h_c = (float *)malloc(n * sizeof(float));

  for (int i = 0; i < n; i++) {
    h_a[i] = rand() / static_cast<float>(RAND_MAX);
    h_a[i] = rand() / static_cast<float>(RAND_MAX);
  }

  float *d_a, *d_b, *d_c;
  CHECK(hipMalloc((void **)&d_a, n * sizeof(float)));
  CHECK(hipMalloc((void **)&d_b, n * sizeof(float)));
  CHECK(hipMalloc((void **)&d_c, n * sizeof(float)));

  CHECK(hipMemcpy(d_a, h_a, n * sizeof(float), hipMemcpyHostToDevice));
  CHECK(hipMemcpy(d_b, h_b, n * sizeof(float), hipMemcpyHostToDevice));

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  hipEventRecord(start);

  int test_rounds = 5;
  for (int i = 0; i < test_rounds; i++) {
    vadd<<<grid_size, block_size>>>(d_a, d_b, d_c, n);
  }

  CHECK(hipMemcpy(h_c, d_c, n * sizeof(float), hipMemcpyDeviceToHost));
  hipEventRecord(stop);

  hipEventSynchronize(stop);
  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);
  printf("The vector add(size %d) time elapsed: %f ms\n", n, milliseconds / float(test_rounds));

  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);

  free(h_a);
  free(h_b);
  free(h_c);
}

void test_max_err(int n) {

  float *h_a = (float *)malloc(n * sizeof(float));
  float *h_b = (float *)malloc(n * sizeof(float));
  float *h_c = (float *)malloc(n * sizeof(float));

  for (int i = 0; i < n; i++) {
    h_a[i] = rand() / static_cast<float>(RAND_MAX);
    h_a[i] = rand() / static_cast<float>(RAND_MAX);
  }

  float *d_a, *d_b, *d_c;
  CHECK(hipMalloc((void **)&d_a, n * sizeof(float)));
  CHECK(hipMalloc((void **)&d_b, n * sizeof(float)));
  CHECK(hipMalloc((void **)&d_c, n * sizeof(float)));

  CHECK(hipMemcpy(d_a, h_a, n * sizeof(float), hipMemcpyHostToDevice));
  CHECK(hipMemcpy(d_b, h_b, n * sizeof(float), hipMemcpyHostToDevice));

  int test_rounds = 5;
  float max_err = 0;
  for (int i = 0; i < test_rounds; i++) {
    vadd<<<(n + 256) / 256, 256>>>(d_a, d_b, d_c, n);
    CHECK(hipMemcpy(h_c, d_c, n * sizeof(float), hipMemcpyDeviceToHost));
    for (int j = 0; j < n; j++) {
      max_err = max(max_err, abs(h_c[j] - (h_a[j] + h_b[j])));
    }
  }

  printf("The vector add(size %d) max err: %f\n", n, max_err);

  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);

  free(h_a);
  free(h_b);
  free(h_c);
}

int main() {
  int nums_n[] = {128, 512, 1024, 2048, 4096};
  int n_size = sizeof(nums_n) / sizeof(int);

  for (int i = 0; i < n_size; i++) {
    test_performance(nums_n[i]);
    test_max_err(nums_n[i]);
  }
}
