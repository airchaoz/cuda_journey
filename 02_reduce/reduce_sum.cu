#include "hip/hip_runtime.h"
#include "bits/stdc++.h"
#include "../helper/error.cuh"

using namespace std;

/**
  *@brief: This function computes the sum of an array
  */
  float kahan_sum_single(const float* arr, int n) {
    float sum = 0.0f;
    float c = 0.0f;
    for (int i = 0; i < n; i++) {
        float y = arr[i] - c;
        float t = sum + y;
        c = (t - sum) - y;
        sum = t;
    }
    return sum;
}

/**
  *@brief: Using shared memory to compute the sum of an array.
  */
__global__ void reduce_sum_v1(float *arr, float *result, int n) {

    extern __shared__ float s_data[];
    int tid = threadIdx.x;
    int idx = blockDim.x * blockIdx.x + tid;

    s_data[tid] = (idx < n) ? arr[idx] : 0.f;
    __syncthreads();

    for (int i = blockDim.x >> 1; i > 0; i = i >> 1) {
        if (tid < i) {
            s_data[tid] += s_data[tid + i];
        }
        __syncthreads();
    }

    if (tid == 0) {
        atomicAdd(result, s_data[0]);
    }
}

void test_error(int n) {

    const int block_size = 128;
    const int grid_size = (n + block_size - 1) / block_size;

    float *h_a = (float *)malloc(n * sizeof(float));
    float *h_result = (float *)malloc(sizeof(float));

    srand(42);
    for (int i = 0; i < n; i++) {
        h_a[i] = rand() / float(RAND_MAX);
    }
    float kahan_result = kahan_sum_single(h_a, n);

    float *d_a;
    float *d_result;

    hipMalloc((void **)&d_a, n * sizeof(float));
    hipMalloc((void **)&d_result, sizeof(float));

    hipMemcpy(d_a, h_a, n * sizeof(float), hipMemcpyHostToDevice);

    reduce_sum_v1<<<grid_size, block_size, block_size * sizeof(float)>>>(d_a, d_result, n);

    hipMemcpy(h_result, d_result, sizeof(float), hipMemcpyDeviceToHost);

    printf("the result of kahan is %f\n", kahan_result);
    printf("the result of v1  is %f\n", *h_result);

    free(h_a);
    free(h_result);

    hipFree(d_result);
    hipFree(d_a);
}

void test_performance(int n) {

    const int block_size = 128;
    const int grid_size = (n + block_size - 1) / block_size;

    float *h_a = (float *)malloc(n * sizeof(float));
    float *h_result = (float *)malloc(grid_size * sizeof(float));

    srand(42);
    for (int i = 0; i < n; i++) {
        h_a[i] = rand() / float(RAND_MAX);
    }

    float *d_a;
    float *d_result;

    hipMalloc((void **)&d_a, n * sizeof(float));
    hipMalloc((void **)&d_result, grid_size * sizeof(float));

    hipMemcpy(d_a, h_a, n * sizeof(float), hipMemcpyHostToDevice);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);

    int test_rounds = 5;
    for (int i = 0; i < test_rounds; i++) {
        reduce_sum_v1<<<grid_size, block_size, block_size * sizeof(float)>>>(d_a, d_result, n);
    }

    hipMemcpy(h_result, d_result, grid_size * sizeof(float), hipMemcpyDeviceToHost);
    hipEventRecord(stop);

    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("The vector reduce_sum_v1(size %d) time elapsed: %f ms\n", n, milliseconds / float(test_rounds));

    free(h_a);
    free(h_result);

    hipFree(d_result);
    hipFree(d_a);
}

int main() {
    const int n = 100'000'000;
    test_error(n);
    test_performance(n);
}