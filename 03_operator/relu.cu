
#include <hip/hip_runtime.h>
#include "bits/stdc++.h"

using namespace std;

#define FLOAT4(x) (reinterpret_cast<float4 *>(&(x))[0])

__global__ void relu_f32(float *x, float *y, int N) {
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    if (tid < N) {
        y[tid] = fmaxf(0.0f, x[tid]);
    }
}

__global__ void relu_f32x4(float *x, float *y, int N) {
    int tid = (blockDim.x * blockIdx.x + threadIdx.x) * 4;
    if (tid < N) {
        float4 reg_a = FLOAT4(x[tid]);

        reg_a.x = fmaxf(reg_a.x, 0.0f);
        reg_a.y = fmaxf(reg_a.y, 0.0f);
        reg_a.z = fmaxf(reg_a.z, 0.0f);
        reg_a.w = fmaxf(reg_a.w, 0.0f);

        FLOAT4(y[tid]) = reg_a;
    }
}